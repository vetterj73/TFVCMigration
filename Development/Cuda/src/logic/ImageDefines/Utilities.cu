/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.
 *
 * This software and the information contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a Non-Disclosure Agreement.  Any reproduction or
 * disclosure to any third party without the express written consent of
 * NVIDIA is prohibited.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.  This source code is a "commercial item" as
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer software" and "commercial computer software
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 */

/* 2D Convolution: C = A (*) B, A is the 5x5 kernel matrix, B is the image matrix.
 * Host code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "GPUManager.h"
#include "GPUStream.h"
#include "../MosaicDataModel/MorphJob.h"

// includes, project
#include <cutil.h>

__constant__ double coeffs[3][3];

// includes, kernels
#include <Utilities_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declarations, forward

//extern "C"
//void computeGold(float*, const float*, const float*, unsigned int, unsigned int);

void CopyBufferToDeviceMatrix(ByteMatrix Mdevice, unsigned char* buffer, hipStream_t *stream);
void CopyDeviceMatrixToHost(ByteMatrix MHost, ByteMatrix Mdevice, hipStream_t *stream);

//ByteMatrix AllocateZeroDeviceMatrix(int width, int height);
//ByteMatrix AllocateByteMatrix(int width, int height);
ByteMatrix AllocateDeviceMatrix(int width, int height);
void ResizeDeviceMatrix(ByteMatrix *Mdevice, int width, int height);
ByteMatrix AllocateHostMatrix(int width, int height);
void ResizeHostMatrix(ByteMatrix *Mhost, int width, int height);
void CopyBufferToDeviceMatrix(ByteMatrix Mdevice, unsigned char* buffer);
void CopyDeviceMatrixToBuffer(ByteMatrix Mdevice, unsigned char* buffer, int hostSpan);
void CopyDeviceMatrixToHost(ByteMatrix MHost, ByteMatrix Mdevice);
void CopyHostMatrixToBuffer(unsigned char* buffer, ByteMatrix Hdevice, int hostSpan);
void FreeDeviceMatrix(ByteMatrix* M);
void FreeMatrix(ByteMatrix* M);
void FreeHostMatrix(ByteMatrix* M);

bool CanMapHostMemory()
{
	struct hipDeviceProp_t prop;

	hipGetDeviceProperties(&prop, 0);
	if (prop.canMapHostMemory) return true;
	else return false;

}
bool CudaBufferRegister(unsigned char *ptr, size_t size)
{
	//hipError_t error = hipHostRegister( ptr, size, hipHostRegisterPortable);
	hipError_t error = hipHostRegister( ptr, size, hipHostRegisterMapped);
 
	if (error != hipSuccess)
		return false;
	
	return true;
}
bool CudaBufferUnregister(unsigned char *ptr)
{
	hipError_t error = hipHostUnregister(ptr);   

	if (error != hipSuccess)
		return false;
	
	return true;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
CyberJob::CGPUJob::GPUJobStatus GPUImageMorph( CyberJob::GPUStream *jobStream,
	unsigned char* pInBuf,  unsigned int iInSpan, 
	unsigned int iInWidth, unsigned int iInHeight, 
	unsigned char* pOutBuf, unsigned int iOutSpan,
	unsigned int iOutROIStartX, unsigned int iOutROIStartY,
	unsigned int iOutROIWidth, unsigned int iOutROIHeight,
	double dInvTrans[3][3]) 
{
	char str[128];
	MorphJob* temp = (MorphJob*)(jobStream->GPUJob());

	unsigned int thePhase = jobStream->Phase();
	jobStream->Phase(thePhase+1);

	switch (thePhase)
	{
	case 0:

		//sprintf_s(str, "Job %d; Phase %d-0;", temp->OrdinalNumber(), thePhase);
		//jobStream->_pGPUJobManager->LogTimeStamp(str);

		//LARGE_INTEGER timestamp;
		///*assert(*/::QueryPerformanceCounter(&timestamp)/*)*/;

		CopyBufferToDeviceMatrix(jobStream->StdInBuffer(), pInBuf, jobStream->Stream());

		//sprintf_s(str, "Job %d; Phase %d; Xfer time", temp->OrdinalNumber(), thePhase);
		//jobStream->_pGPUJobManager->DeltaTimeStamp(str, timestamp);

		// Copy coefficients to device constant memory
		hipMemcpyToSymbolAsync(HIP_SYMBOL(coeffs), dInvTrans, sizeof(dInvTrans[0])*3, 0, hipMemcpyHostToDevice, *jobStream->Stream());

		//for (int j=0; j<1/*8*/; ++j)
		{

			// Setup the execution configuration
			dim3 threads(TILE_WIDTH, 12/*TILE_WIDTH*/);
			dim3 grid(((iOutROIWidth - 1) / threads.x) + 1, ((iOutROIHeight - 1) / threads.y) + 1);

  			// Launch the device computation threads!
			ConvolutionKernel<<< grid, threads, 0, *jobStream->Stream()>>>
				(jobStream->StdInBuffer().elements, jobStream->StdOutBuffer().elements, iInSpan,
				iInHeight, iInWidth,
				iOutROIWidth, iOutROIHeight, iOutSpan, iOutROIStartX, iOutROIStartY);
		}

		return CGPUJob::GPUJobStatus::ACTIVE;

	case 1:
		//CopyDeviceMatrixToHost(context->B, context->Bd, jobStream->Stream());

		/*hipError_t */hipMemcpy2D  ( pOutBuf + iOutROIStartX + iOutSpan * iOutROIStartY,  
				iOutSpan,
				jobStream->StdOutBuffer().elements,
				iOutROIWidth,
				iOutROIWidth,
				iOutROIHeight,
				hipMemcpyDeviceToHost );

		hipEventCreate(jobStream->PhaseEvent());

		hipEventRecord(*jobStream->PhaseEvent(), *jobStream->Stream());

		return CGPUJob::GPUJobStatus::ACTIVE;

	case 2:
		//hipError_t result = hipEventQuery(context->phaseEvent);
		//if (result != hipSuccess)
		//{
		//	if (result == hipErrorNotReady)
		//	{
		//		//sprintf_s(str, "Job %d; Phase %d; hipErrorNotReady", temp->OrdinalNumber(), thePhase);
		//		//jobStream->_pGPUJobManager->LogTimeStamp(str);
		//	}
		//	else
		//	{
		//		sprintf_s(str, "Job %d; Phase %d; hipError_t %d;", temp->OrdinalNumber(), thePhase, result);
		//		//jobStream->_pGPUJobManager->LogTimeStamp(str);
		//	}

		//	// maintain current phase to continue to check CopyDeviceMatrixToHost event for completion
		//	jobStream->Phase(thePhase);

		//	return CGPUJob::GPUJobStatus::WAITING;
		//}

		hipEventSynchronize(*jobStream->PhaseEvent()/*context->phaseEvent*/); // wait on CopyDeviceMatrixToHost event not used

		// copy morphed FOV image to panel image buffer
		//CopyHostMatrixToBuffer(pOutBuf + iOutROIStartX + iOutSpan * iOutROIStartY, context->B, iOutSpan);

		hipEventDestroy(*jobStream->PhaseEvent()/*context->phaseEvent*/);

		return CGPUJob::GPUJobStatus::COMPLETED;
	}

	return CGPUJob::GPUJobStatus::COMPLETED;
}

// Allocate a device matrix of same size as M.
ByteMatrix AllocateDeviceMatrix(int width, int height)
{
	ByteMatrix Mdevice;

    Mdevice.width = width;
    Mdevice.height = height;
    Mdevice.size = width * height * sizeof(unsigned char);
    Mdevice.elements = NULL;

    hipMalloc((void**)&Mdevice.elements, Mdevice.size);

	return Mdevice;
}

void ResizeDeviceMatrix(ByteMatrix *Mdevice, int width, int height)
{
	if (width*height > Mdevice->size)
	{
		if (Mdevice->elements != NULL) hipFree(Mdevice->elements);
		Mdevice->elements = NULL;
		hipMalloc( &Mdevice->elements, width*height*sizeof(unsigned char));
		Mdevice->size = width*height;
	}
    Mdevice->width = width;
    Mdevice->height = height;
}

// Allocate a host matrix of dimensions height*width
ByteMatrix AllocateHostMatrix(int width, int height)
{
	ByteMatrix Mhost;

    Mhost.width = width;
    Mhost.height = height;
    Mhost.size = width * height * sizeof(unsigned char);

	Mhost.elements = NULL;
	hipHostMalloc( &Mhost.elements, Mhost.size*sizeof(unsigned char));
	//Mhost.elements = (unsigned char*) malloc(Mhost.size*sizeof(unsigned char));

	return Mhost;
}

void ResizeHostMatrix(ByteMatrix *Mhost, int width, int height)
{
	if (width*height > Mhost->size)
	{
		if (Mhost->elements != NULL)
		{
			 hipError_t error = hipHostFree(Mhost->elements);
			 //delete Mhost.elements;
		}
		Mhost->elements = NULL;
		hipHostMalloc( &Mhost->elements, width*height*sizeof(unsigned char));
		//Mhost.elements = (unsigned char*) malloc(Mhost.size*sizeof(unsigned char));
		Mhost->size = width*height;
	}
    Mhost->width = width;
    Mhost->height = height;
}

//// Allocate a host matrix of dimensions height*width
//ByteMatrix AllocateByteMatrix(int width, int height)
//{
//    ByteMatrix M;
//    M.width = M.pitch = width;
//    M.height = height;
//    int size = M.width * M.height;
//
//	M.elements = NULL;
//	M.elements = (unsigned char*) malloc(size*sizeof(unsigned char));
//
//    return M;
//}
//
// Copy a host matrix to a device matrix.
void CopyBufferToDeviceMatrix(ByteMatrix Mdevice, unsigned char* buffer, hipStream_t *stream)
{
    int size = Mdevice.width * Mdevice.height * sizeof(unsigned char);
    hipMemcpyAsync(Mdevice.elements, buffer, size, hipMemcpyHostToDevice, *stream);
}
// Copy a host matrix to a device matrix.
void CopyBufferToDeviceMatrix(ByteMatrix Mdevice, unsigned char* buffer)
{
    int size = Mdevice.width * Mdevice.height * sizeof(unsigned char);
    hipMemcpy(Mdevice.elements, buffer, size, hipMemcpyHostToDevice);
}

void CopyDeviceMatrixToHost(ByteMatrix Mhost, ByteMatrix Mdevice, hipStream_t *stream)
{
    int Hsize = Mhost.width * Mhost.height * sizeof(unsigned char);
    int Dsize = Mdevice.width * Mdevice.height * sizeof(unsigned char);
	int size = (Dsize > Hsize) ? Hsize : Dsize ;

	hipMemcpyAsync(Mhost.elements, Mdevice.elements, size, hipMemcpyDeviceToHost, *stream);
}
void CopyDeviceMatrixToHost(ByteMatrix Mhost, ByteMatrix Mdevice)
{
    int Hsize = Mhost.width * Mhost.height * sizeof(unsigned char);
    int Dsize = Mdevice.width * Mdevice.height * sizeof(unsigned char);
	int size = (Dsize > Hsize) ? Hsize : Dsize ;

	hipMemcpy(Mhost.elements, Mdevice.elements, size, hipMemcpyDeviceToHost);
}

void CopyHostMatrixToBuffer(unsigned char* buffer, ByteMatrix Hdevice, int hostSpan)
{
	for (int i=0; i<Hdevice.height; ++i)
	{
		memcpy(buffer+i*hostSpan, Hdevice.elements+i*Hdevice.width, Hdevice.width);
	}
}

// Free a device matrix.
void FreeDeviceMatrix(ByteMatrix* M)
{
    hipFree(M->elements);
    M->elements = NULL;
}

// Free a host ByteMatrix
void FreeHostMatrix(ByteMatrix* M)
{
    hipHostFree(M->elements);
    M->elements = NULL;
}
