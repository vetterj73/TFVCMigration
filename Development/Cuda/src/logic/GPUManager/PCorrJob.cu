#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "GPUManager.h"
#include "GPUStream.h"

// includes, project
#include <cutil.h>
#include <hipfft/hipfft.h>

typedef struct {
   float r;
   float i;
} complexf;

//__constant__ double coeffs[3][3];
__constant__ float acurve[256];
__constant__ float bcurve[256];
__constant__ float dKernel[10];

// includes, kernels
#include <PCorrJob_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declarations, forward

//extern "C"
//void computeGold(float*, const float*, const float*, unsigned int, unsigned int);

//void CopyBufferToDeviceMatrix(ByteMatrix Mdevice, unsigned char* buffer, hipStream_t *stream);
//void CopyDeviceMatrixToHost(ByteMatrix MHost, ByteMatrix Mdevice, hipStream_t *stream);
//
////ByteMatrix AllocateZeroDeviceMatrix(int width, int height);
////ByteMatrix AllocateByteMatrix(int width, int height);
//ByteMatrix AllocateDeviceMatrix(int width, int height);
//void ResizeDeviceMatrix(ByteMatrix *Mdevice, int width, int height);
//ByteMatrix AllocateHostMatrix(int width, int height);
//void ResizeHostMatrix(ByteMatrix *Mhost, int width, int height);
//void CopyBufferToDeviceMatrix(ByteMatrix Mdevice, unsigned char* buffer);
//void CopyDeviceMatrixToBuffer(ByteMatrix Mdevice, unsigned char* buffer, int hostSpan);
//void CopyDeviceMatrixToHost(ByteMatrix MHost, ByteMatrix Mdevice);
//void CopyHostMatrixToBuffer(unsigned char* buffer, ByteMatrix Hdevice, int hostSpan);
//void FreeDeviceMatrix(ByteMatrix* M);
//void FreeMatrix(ByteMatrix* M);
//void FreeHostMatrix(ByteMatrix* M);

//bool CanMapHostMemory()
//{
//	struct hipDeviceProp_t prop;
//
//	hipGetDeviceProperties(&prop, 0);
//	if (prop.canMapHostMemory) return true;
//	else return false;
//
//}
//bool CudaBufferRegister(unsigned char *ptr, size_t size)
//{
//	hipError_t error = hipHostRegister( ptr, size, hipHostRegisterPortable);
//	//hipError_t error = hipHostRegister( ptr, size, hipHostRegisterMapped);
// 
//	if (error != hipSuccess)
//		return false;
//	
//	return true;
//}
//bool CudaBufferUnregister(unsigned char *ptr)
//{
//	hipError_t error = hipHostUnregister(ptr);   
//
//	if (error != hipSuccess)
//		return false;
//	
//	return true;
//}
//
//


static const float kernel2coef[] = {
	+6.913469554E-01,   /*  0.5 */
	+3.963233672E-01,   /*  1.5 */
	+5.312047567E-02,   /*  2.5 */
	-1.083921551E-01,   /*  3.5 */
	-8.447095014E-02    /*  4.5 */
};
static const float kernel4coef[] = {
	+3.658160711E-01,   /*  0.5 */
	+3.215795429E-01,   /*  1.5 */
	+2.437565548E-01,   /*  2.5 */
	+1.502194757E-01,   /*  3.5 */
	+6.090930440E-02,   /*  4.5 */
	-8.232239494E-03,   /*  5.5 */
	-4.850434474E-02,   /*  6.5 */
	-6.070450677E-02,   /*  7.5 */
	-5.144587161E-02,   /*  8.5 */
	-3.450495203E-02    /*  9.5 */
};

void GPUPCorrExit()
{
	hipError_t temp = hipDeviceReset();
}

CyberGPU::CGPUJob::GPUJobStatus GPUPCorr( CyberGPU::GPUStream *jobStream,
	int ncols,			/* Number of columns in images */
	int nrows,			/* Number of rows in images */
	unsigned char a[],	/* Pointer to first image  */
	unsigned char b[],	/* Pointer to second image */
	int astride,
	int bstride,
	float apal[], float bpal[],
	/*int columns, int rows,*/ int decimx, int decimy,
	int ncd, int nrd, complexf * z, float * work, int crosswindow, hipfftHandle plan) 
{
	//hipfftHandle plan;
	hipfftResult results;

	unsigned int thePhase = jobStream->Phase();
	jobStream->Phase(thePhase+1);

	int size = ncd * nrd * sizeof(complexf);

	switch (thePhase)
	{
	case 0:
		//result = hipfftPlan2d( jobStream->Plan(), nrd, ncd, HIPFFT_C2C);
		results = hipfftSetStream(plan, *jobStream->Stream());
		if (results != HIPFFT_SUCCESS)
		{
			results = (hipfftResult)0; // code to break on
			// log error
		}
		//CopyBufferToDeviceMatrix(jobStream->StdInBuffer(), pInBuf, jobStream->Stream());

		////sprintf_s(str, "Job %d; Phase %d; Xfer time", temp->OrdinalNumber(), thePhase);
		////jobStream->_pGPUJobManager->DeltaTimeStamp(str, timestamp);

		{

			// copy A to device stdin beginning
			//hipMemcpyAsync(jobStream->StdInBuffer().elements, a/*(unsigned char*)z*/,
			//	imagesize, hipMemcpyHostToDevice, *jobStream->Stream());

			hipError_t error2D = hipMemcpy2DAsync(
				jobStream->StdInBuffer().elements,  
				ncols,
				a,
				astride,
				ncols,
				nrows,
				hipMemcpyHostToDevice,
				 *jobStream->Stream());
			if (error2D != hipSuccess)
			{
				error2D = (hipError_t)0; // code to break on
				// log error
			}
			

			// copy B to device stdin with offset
			int alignment = 0x20;
			int offset = ncols * nrows * sizeof(unsigned char);
			if (offset % alignment ) offset += alignment - (offset%alignment);
			//hipMemcpyAsync(jobStream->StdInBuffer().elements+offset,
			//	b, imagesize, hipMemcpyHostToDevice, *jobStream->Stream());
			error2D = hipMemcpy2DAsync(
				jobStream->StdInBuffer().elements+offset,  
				ncols,
				b,
				bstride,
				ncols,
				nrows,
				hipMemcpyHostToDevice,
				 *jobStream->Stream());
			if (error2D != hipSuccess)
			{
				error2D = (hipError_t)0; // code to break on
				// log error
			}

			// copy apal to device constant memory
			hipMemcpyToSymbolAsync(HIP_SYMBOL(acurve), apal, 256*sizeof(float), 0, hipMemcpyHostToDevice, *jobStream->Stream());
			// copy bpal to device constant memory
			hipMemcpyToSymbolAsync(HIP_SYMBOL(bcurve), bpal, 256*sizeof(float), 0, hipMemcpyHostToDevice, *jobStream->Stream());

			// Setup merge of a and b images into complex image
			dim3 threads(TILE_WIDTH, /*12*/TILE_WIDTH);
			dim3 grid(((ncols - 1) / threads.x) + 1, ((nrows - 1) / threads.y) + 1);

			ApplyEqualizationKernel<<< grid, threads, 0, *jobStream->Stream()>>>
				(jobStream->StdInBuffer().elements,
				jobStream->StdInBuffer().elements+offset,
				(complexf*)jobStream->StdOutBuffer().elements,
				ncols, nrows,
				ncols, ncols);

#ifdef __TEXTURE_COEF
			hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

			hipArray* cuArray;
			hipMallocArray(&cuArray, &channelDesc, 10, 1);
			
			// Set texture parameters
			CyberGPU::texKernel.addressMode[0] = hipAddressModeClamp;
			CyberGPU::texKernel.addressMode[1] = hipAddressModeClamp;
			CyberGPU::texKernel.filterMode = hipFilterModePoint;
			CyberGPU::texKernel.normalized = false;
#endif

			bool decimate = true;
			unsigned int decimx_mask = 0;
			switch (decimx)
			{
			case 2:
				hipMemcpyToSymbol(HIP_SYMBOL(dKernel), kernel2coef, sizeof(kernel2coef), 0, hipMemcpyHostToDevice);
				decimx_mask = 1;
#ifdef __TEXTURE_COEF
				hipMemcpyToArray(cuArray, 0, 0, kernel2coef, sizeof(kernel2coef), hipMemcpyHostToDevice);
#else
				hipMemcpyToSymbol(HIP_SYMBOL(dKernel), kernel2coef, sizeof(kernel2coef), 0, hipMemcpyHostToDevice);
#endif
			
				break;
			case 4:
				decimx_mask = 3;
#ifdef __TEXTURE_COEF
				hipMemcpyToArray(cuArray, 0, 0, kernel4coef, sizeof(kernel4coef), hipMemcpyHostToDevice);
#else
				hipMemcpyToSymbol(HIP_SYMBOL(dKernel), kernel4coef, sizeof(kernel4coef), 0, hipMemcpyHostToDevice);
#endif
				break;
			default:
				decimate = false;
				break;
			}

			if (decimate)
			{
#ifdef __TEXTURE_COEF
				// Bind the array to the texture reference
				hipBindTextureToArray(CyberGPU::texKernel, cuArray, channelDesc);
#endif

				// !!! if ncol (the number of undecimated columns is greater than 500, shared memory is not big enough
				//     dynamically created shared memory can be passed as the third argument in a cuda call. This is
				//     one solution. Hard coded shared memory can be allocated inside the cuda call. This is the current
				//     solution to get 512*sizeof(complexf) bytes allocated for shared memory

				// Setup horizontal image decimation
				dim3 Hthreads(ncd, 1); // a block is a decimated row
				dim3 Hgrid(1, nrows); // grid is undecimated number of rows 

  				// Launch the device computation threads!
				DecimHorizontalKernel<<< Hgrid, Hthreads, (ncols+16)*sizeof(complexf), *jobStream->Stream()>>>
					((complexf*)jobStream->StdOutBuffer().elements,
					(complexf*)jobStream->StdInBuffer().elements,
					ncols, nrows, ncols, ncd, decimx, 2*decimx + decimx/2 ); // 2*decimx + decimx/2; // 2->5, 4->10

				//dim3 threads(TILE_16, TILE_16);
				//dim3 grid(((ncols - TILE_16 - 1) / threads.x) + 1, ((nrows - 1) / threads.y) + 1);

				//DecimHorizontalKernelInside<<< grid, threads, ncd, *jobStream->Stream()>>>
				//	((complexf*)jobStream->StdOutBuffer().elements,
				//	(complexf*)jobStream->StdInBuffer().elements,
				//	ncols, nrows, ncols, ncd, decimx, decimx_mask/*2*decimx + decimx/2*/ ); // 2*decimx + decimx/2; // 2->5, 4->10
			}

			if (decimate)
			{
				dim3 Tthreads(TILE_DIM, TILE_DIM);
				dim3 Tgrid(((ncd - 1) / Tthreads.x) + 1, ((nrows - 1) / Tthreads.y) + 1);

  				// Launch the device computation threads!
				TransposeMatrix<<< Tgrid, Tthreads, 0, *jobStream->Stream()>>>
					((complexf*)jobStream->StdInBuffer().elements,
					(complexf*)jobStream->StdOutBuffer().elements,
					ncd, nrows );
			}

			decimate = true;
			switch (decimy)
			{
			case 2:
				hipMemcpyToSymbol(HIP_SYMBOL(dKernel), kernel2coef, sizeof(kernel2coef), 0, hipMemcpyHostToDevice);
				break;
			case 4:
				hipMemcpyToSymbol(HIP_SYMBOL(dKernel), kernel4coef, sizeof(kernel4coef), 0, hipMemcpyHostToDevice);
				break;
			default:
				decimate = false;
				break;
			}

			if (decimate)
			{
				// Setup vertical image decimation
				dim3 Vthreads(1, nrd); // a block is a decimated column
				//dim3 Vgrid(ncols, 1); // grid is decimated number of columns 
				dim3 Vgrid(ncd, 1); // grid is decimated number of columns 

  				// Launch the device computation threads!
				DecimVerticalKernel<<< Vgrid, Vthreads, (nrows+16)*sizeof(complexf), *jobStream->Stream()>>>
					((complexf*)jobStream->StdOutBuffer().elements,
					(complexf*)jobStream->StdInBuffer().elements,
					//((complexf*)jobStream->StdInBuffer().elements,
					//(complexf*)jobStream->StdOutBuffer().elements,
					//nrows, ncols, ncols, decimy, 2*decimy + decimy/2 ); // 2*decimx + decimx/2; // 2->5, 4->10
					nrows, /*ncd*/nrows, ncd, decimy, 2*decimy + decimy/2 ); // 2*decimx + decimx/2; // 2->5, 4->10
			}
		}

		results = hipfftExecC2C( plan, (hipfftComplex*)jobStream->StdInBuffer().elements,
			(hipfftComplex*)jobStream->StdInBuffer().elements, HIPFFT_FORWARD);
		if (results != HIPFFT_SUCCESS)
		{
			hipError_t err = hipGetLastError();
			// log error
		}

		{
			// Setup vertical circular convolution for CrossFilter
			dim3 Vthreads(1, nrd); // 
			dim3 Vgrid(crosswindow*2 + 1, 1);

  			// Launch the device computation threads!
			CrossFilterVerticalKernel<<< Vgrid, Vthreads, (nrd+2)*sizeof(complexf), *jobStream->Stream()>>>
				((complexf*)jobStream->StdInBuffer().elements,
				ncd, nrd, crosswindow, ncd);

			// Setup horizontal circular convolution for CrossFilter
			dim3 Hthreads(ncd, 1);
			dim3 Hgrid(1, crosswindow*2 + 1);

  			// Launch the device computation threads!
			CrossFilterHorizontalKernel<<< Hgrid, Hthreads, (ncd+2)*sizeof(complexf), *jobStream->Stream()>>>
				((complexf*)jobStream->StdInBuffer().elements,
				ncd, nrd, crosswindow, ncd);
		}

		{

			int alignment = 0x20;
			int offset = ncols * nrows * sizeof(complexf);
			if (offset % alignment ) offset += alignment - (offset%alignment);

			// Setup conjugate multiplication
			dim3 threads(TILE_WIDTH, /*12*/TILE_WIDTH);
			dim3 grid(((ncd - 1) / threads.x) + 1, ((nrd/2/* - 1*/) / threads.y) + 1);

  			// Launch the device computation threads!
			ConjugateMultKernel<<< grid, threads, 0, *jobStream->Stream()>>>
				((complexf*)jobStream->StdInBuffer().elements,
				(complexf*)jobStream->StdOutBuffer().elements,
				(float*)(jobStream->StdInBuffer().elements+offset),
				ncd, nrd, ncd);

		}

		results = hipfftExecC2C( plan, (hipfftComplex*)jobStream->StdOutBuffer().elements,
			(hipfftComplex*)jobStream->StdOutBuffer().elements, HIPFFT_BACKWARD);
		if (results != HIPFFT_SUCCESS)
		{
			hipError_t err = hipGetLastError();
			// log error
		}

		return CyberGPU::CGPUJob::GPUJobStatus::ACTIVE;

	case 1:
		{
			int alignment = 0x20;
			int offset = ncols * nrows * sizeof(complexf);
			if (offset % alignment ) offset += alignment - (offset%alignment);
			int total = (((ncd - 1) / TILE_WIDTH) + 1) * (((nrd/2/* - 1*/) / TILE_WIDTH) + 1);
			hipError_t error = hipMemcpyAsync(work, jobStream->StdInBuffer().elements+offset,
				total*sizeof(float), hipMemcpyDeviceToHost, *jobStream->Stream());

			int imagesize = /*110*/ncd * /*378*/nrd * sizeof(complexf);
			//int imagesize = ncols * nrows * sizeof(complexf);
			//error = hipMemcpyAsync(z, jobStream->StdInBuffer().elements,
			error = hipMemcpyAsync(z, jobStream->StdOutBuffer().elements,
				imagesize, hipMemcpyDeviceToHost, *jobStream->Stream());
		}

		//hipEventDestroy(*jobStream->PhaseEvent());

		hipEventCreate(jobStream->PhaseEvent());

		hipEventRecord(*jobStream->PhaseEvent(), *jobStream->Stream());

		return CyberGPU::CGPUJob::GPUJobStatus::ACTIVE;
	case 2:
		//hipError_t result = hipEventQuery(context->phaseEvent);
		//if (result != hipSuccess)
		//{
		//	if (result == hipErrorNotReady)
		//	{
		//		//sprintf_s(str, "Job %d; Phase %d; hipErrorNotReady", temp->OrdinalNumber(), thePhase);
		//		//jobStream->_pGPUJobManager->LogTimeStamp(str);
		//	}
		//	else
		//	{
		//		sprintf_s(str, "Job %d; Phase %d; hipError_t %d;", temp->OrdinalNumber(), thePhase, result);
		//		//jobStream->_pGPUJobManager->LogTimeStamp(str);
		//	}

		//	// maintain current phase to continue to check CopyDeviceMatrixToHost event for completion
		//	jobStream->Phase(thePhase);

		//	return CGPUJob::GPUJobStatus::WAITING;
		//}

		hipEventSynchronize(*jobStream->PhaseEvent()/*context->phaseEvent*/); // wait on CopyDeviceMatrixToHost event not used

		// copy morphed FOV image to panel image buffer
		//CopyHostMatrixToBuffer(pOutBuf + iOutROIStartX + iOutSpan * iOutROIStartY, context->B, iOutSpan);

		hipEventDestroy(*jobStream->PhaseEvent()/*context->phaseEvent*/);

		return CyberGPU::CGPUJob::GPUJobStatus::COMPLETED;
	}

	return CyberGPU::CGPUJob::GPUJobStatus::COMPLETED;
}
//// Allocate a device matrix of same size as M.
//ByteMatrix AllocateDeviceMatrix(int width, int height)
//{
//	ByteMatrix Mdevice;
//
//    Mdevice.width = width;
//    Mdevice.height = height;
//    Mdevice.size = width * height * sizeof(unsigned char);
//    Mdevice.elements = NULL;
//
//    hipMalloc((void**)&Mdevice.elements, Mdevice.size);
//
//	return Mdevice;
//}
//
//void ResizeDeviceMatrix(ByteMatrix *Mdevice, int width, int height)
//{
//	if (width*height > Mdevice->size)
//	{
//		if (Mdevice->elements != NULL) hipFree(Mdevice->elements);
//		Mdevice->elements = NULL;
//		hipMalloc( &Mdevice->elements, width*height*sizeof(unsigned char));
//		Mdevice->size = width*height;
//	}
//    Mdevice->width = width;
//    Mdevice->height = height;
//}
//
//// Allocate a host matrix of dimensions height*width
//ByteMatrix AllocateHostMatrix(int width, int height)
//{
//	ByteMatrix Mhost;
//
//    Mhost.width = width;
//    Mhost.height = height;
//    Mhost.size = width * height * sizeof(unsigned char);
//
//	Mhost.elements = NULL;
//	hipHostMalloc( &Mhost.elements, Mhost.size*sizeof(unsigned char));
//	//Mhost.elements = (unsigned char*) malloc(Mhost.size*sizeof(unsigned char));
//
//	return Mhost;
//}
//
//void ResizeHostMatrix(ByteMatrix *Mhost, int width, int height)
//{
//	if (width*height > Mhost->size)
//	{
//		if (Mhost->elements != NULL)
//		{
//			 hipError_t error = hipHostFree(Mhost->elements);
//			 //delete Mhost.elements;
//		}
//		Mhost->elements = NULL;
//		hipHostMalloc( &Mhost->elements, width*height*sizeof(unsigned char));
//		//Mhost.elements = (unsigned char*) malloc(Mhost.size*sizeof(unsigned char));
//		Mhost->size = width*height;
//	}
//    Mhost->width = width;
//    Mhost->height = height;
//}
//
//// Copy a host matrix to a device matrix.
//void CopyBufferToDeviceMatrix(ByteMatrix Mdevice, unsigned char* buffer, hipStream_t *stream)
//{
//    int size = Mdevice.width * Mdevice.height * sizeof(unsigned char);
//    hipMemcpyAsync(Mdevice.elements, buffer, size, hipMemcpyHostToDevice, *stream);
//}
//// Copy a host matrix to a device matrix.
//void CopyBufferToDeviceMatrix(ByteMatrix Mdevice, unsigned char* buffer)
//{
//    int size = Mdevice.width * Mdevice.height * sizeof(unsigned char);
//    hipMemcpy(Mdevice.elements, buffer, size, hipMemcpyHostToDevice);
//}
//
//void CopyDeviceMatrixToHost(ByteMatrix Mhost, ByteMatrix Mdevice, hipStream_t *stream)
//{
//    int Hsize = Mhost.width * Mhost.height * sizeof(unsigned char);
//    int Dsize = Mdevice.width * Mdevice.height * sizeof(unsigned char);
//	int size = (Dsize > Hsize) ? Hsize : Dsize ;
//
//	hipMemcpyAsync(Mhost.elements, Mdevice.elements, size, hipMemcpyDeviceToHost, *stream);
//}
//void CopyDeviceMatrixToHost(ByteMatrix Mhost, ByteMatrix Mdevice)
//{
//    int Hsize = Mhost.width * Mhost.height * sizeof(unsigned char);
//    int Dsize = Mdevice.width * Mdevice.height * sizeof(unsigned char);
//	int size = (Dsize > Hsize) ? Hsize : Dsize ;
//
//	hipMemcpy(Mhost.elements, Mdevice.elements, size, hipMemcpyDeviceToHost);
//}
//
//void CopyHostMatrixToBuffer(unsigned char* buffer, ByteMatrix Hdevice, int hostSpan)
//{
//	for (int i=0; i<Hdevice.height; ++i)
//	{
//		memcpy(buffer+i*hostSpan, Hdevice.elements+i*Hdevice.width, Hdevice.width);
//	}
//}
//
//// Free a device matrix.
//void FreeDeviceMatrix(ByteMatrix* M)
//{
//    hipFree(M->elements);
//    M->elements = NULL;
//}
//
//// Free a host ByteMatrix
//void FreeHostMatrix(ByteMatrix* M)
//{
//    hipHostFree(M->elements);
//    M->elements = NULL;
//}
